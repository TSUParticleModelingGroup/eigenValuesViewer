#include "hip/hip_runtime.h"
//nvcc EigenValueViewerBells.cu -o bells -lglut -lm -lGLU -lGL
//#include <GL/glut.h>
//#include <math.h>
//#include <stdio.h>
//#include <device.h>

#include <iostream>
#include <fstream>
#include <sstream>
#include <string.h>
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
using namespace std;

#define SCALE 50.0

#define X_WINDOW 1000
#define Y_WINDOW 1000

#define X_MAX SCALE
#define X_MIN -SCALE
#define X_SCALE 1.0

#define Y_MAX SCALE
#define Y_MIN -SCALE
#define Y_SCALE 1.0

FILE* ffmpeg;

// function prototypes
void KeyPressed(unsigned char key, int x, int y);
void Display(void);

//globalsgcc FunctionHit.c -o FunctionHit -lglut -lm -lGLU -lGL
double g_x;
double g_y;
static int g_win;
double A11, A12, A21, A22;
int MovieOn = 0;
int* Buffer;

double x_machine_to_x_screen(int x)
{
	return( (2.0*x)/X_WINDOW-1.0 );
}

double y_machine_to_y_screen(int y)
{
	return( -(2.0*y)/Y_WINDOW+1.0 );
}

double x_machine_to_x_world(int x)
{
	double range;
	range = X_MAX - X_MIN;
	return( (range/X_WINDOW)*x + X_MIN);
}

double y_machine_to_y_world(int y)
{
	double range;
	range = Y_MAX - Y_MIN;
	return(-((range/Y_WINDOW)*y - X_MAX));
}

double x_world_to_x_screen(double x)
{
	double range;
	range = X_MAX - X_MIN;
	return( -1.0 + 2.0*(x - X_MIN)/range );
}

double y_world_to_y_screen(double y)
{
	double range;
	range = Y_MAX - Y_MIN;
	return( -1.0 + 2.0*(y - Y_MIN)/range );
}

void place_axis()
{
	glColor3f(1.0,1.0,1.0);

	glBegin(GL_LINE_LOOP);
		glVertex2f(x_machine_to_x_screen(0),0);
		glVertex2f(x_machine_to_x_screen(X_WINDOW),0);
	glEnd();

	glBegin(GL_LINE_LOOP);
		glVertex2f(0,y_machine_to_y_screen(0));
		glVertex2f(0,y_machine_to_y_screen(Y_WINDOW));
	glEnd();

	glFlush();
}

void placePoint(double x, double y)
{
	glPointSize(5);
	glBegin(GL_POINTS);
		glVertex2f(x_world_to_x_screen(x),y_world_to_y_screen(y));
	glEnd();
	glFlush();
	
	if(MovieOn == 1)
	{
		glReadPixels(5, 5, X_WINDOW, Y_WINDOW, GL_RGBA, GL_UNSIGNED_BYTE, Buffer);
		fwrite(Buffer, sizeof(int)*X_WINDOW*Y_WINDOW, 1, ffmpeg);
	}
}

void hitMatrix(double x, double y)
{
	double xOld = x;
	double yOld = y;
	
	g_x = A11*xOld + A12*yOld;
	g_y = A21*xOld + A22*yOld;
}

void printPoint()
{
	printf("\n  x = %f\n",g_x);
	printf("  y = %f\n",g_y);
}

void mymouse(int button, int state, int x, int y)
{	
	if(state == GLUT_DOWN)
	{
		if(button == GLUT_LEFT_BUTTON)
		{
			glColor3f(1.0,1.0,0.0);
			g_x = x_machine_to_x_world(x);
			g_y = y_machine_to_y_world(y);
			placePoint(g_x,g_y);
			printPoint();
		}
		else
		{
			glColor3f(0.0,1.0,0.0);
			hitMatrix(g_x,g_y);
			placePoint(g_x,g_y);
			printPoint();
		}
	}
}

void KeyPressed(unsigned char key, int x, int y)
{
	float tempx;
	float tempy;
	float damp;
	if(key == 'q')
	{
		glutDestroyWindow(g_win);
		exit(0);
	}
	if(key == 't')
	{
		printf("\n  Inter start x value\n");
		scanf("%f", &tempx);
		g_x = tempx;
		printf("\n  Inter start y value\n");
		scanf("%f", &tempy);
		g_y = tempy;
		
		glColor3f(1.0,0.0,0.0);
		placePoint(g_x,g_y);
		printPoint();
	}
	if(key == 'c')
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
	}
	
	if(key == 'm')
	{
		// Setting up the movie buffer.
		const char* cmd = "ffmpeg -r 60 -f rawvideo -pix_fmt rgba -s 1000x1000 -i - "
		              "-threads 0 -preset fast -y -pix_fmt yuv420p -crf 21 -vf vflip output.mp4";
		ffmpeg = popen(cmd, "w");
		//Buffer = new int[XWindowSize*YWindowSize];
		Buffer = (int*)malloc(X_WINDOW*Y_WINDOW*sizeof(int));
		MovieOn = 1;
	}
	if(key == 'M')
	{
		pclose(ffmpeg);
		MovieOn = 0;
	}
	
	if(key == 's')
	{	
		FILE* ScreenShotFile;
		int* buffer;
		const char* cmd = "ffmpeg -r 60 -f rawvideo -pix_fmt rgba -s 1000x1000 -i - "
		              "-threads 0 -preset fast -y -pix_fmt yuv420p -crf 21 -vf vflip output1.mp4";
		ScreenShotFile = popen(cmd, "w");
		buffer = (int*)malloc(X_WINDOW*Y_WINDOW*sizeof(int));
		
		for(int i =0; i < 1; i++)
		{
			glReadPixels(5, 5, X_WINDOW, Y_WINDOW, GL_RGBA, GL_UNSIGNED_BYTE, buffer);
			fwrite(buffer, sizeof(int)*X_WINDOW*Y_WINDOW, 1, ScreenShotFile);
		}
		
		pclose(ScreenShotFile);
		free(buffer);
		system("ffmpeg -i output1.mp4 screenShot.jpeg");
		system("rm output1.mp4");
	}
	
	if(key == 'h')
	{
		printf("\n q: Quit");
		printf("\n c: Clear");
		printf("\n M/m: Movie on/off");
		printf("\n s: Screen Shot");
		printf("\n");
		printf("\n t: Read starting x and y from screen.");
		printf("\n 0: Eigenvalue 5/4 eigenvector <2,1> Eigenvalue 1/2 eigenvector <1,-1>");
		printf("\n 1: Eigenvalue 3/2 eigenvector <1,1> Eigenvalue -1/2 eigenvector <1,-1>");
		printf("\n 2: Eigenvalue sqrt(3)/2 eigenvector <sqrt(3)+1,1> Eigenvalue -sqrt(3)/2 eigenvector <-sqrt(3)+1,1> equal size eigen values one negative makes it alternate");
		printf("\n 3: Eigenvalue 1.1 eigenvector <1,0> Eigenvalue 1.2 eigenvector <0,1>");
		printf("\n 4: Eigenvalue 1 eigenvector <1,0> Eigenvalue 1.2 <0,1>");
		printf("\n 5: Eigenvalue 1.1 eigenvector <1,1> Eigenvalue 1.2 eigenvector <0,1>");
		printf("\n 6: Eigenvalue -0.5 eigenvector <-3,4> Eigenvalue 1.25 eigenvector <1,1>");
		printf("\n 7: Eigenvalue i eigenvector <-2-i,5> Eigenvalue -i eigenvector <-2+i,5>  spiral 4 cycle");
		printf("\n 8: Eigenvalue 1/2 + i*sqrt(2)/2 eigenvector <i*sqrt(2),1> Eigenvalue 1/2 - i*sqrt(2)/2 eigenvector <-i*sqrt(2),1>  spiral in");
		printf("\n 9: Eigenvalue damp*(1+i*sqrt(2)) eigenvector <i*sqrt(2),1> Eigenvalue damp*(1-i*sqrt(2)) eigenvector <-i*sqrt(2),1>  spiral out");
		printf("\n");
	}
	
	if(key == '0') // Eigenvalue 5/4 eigenvector <2,1> Eigenvalue 1/2 eigenvector <1,-1> .2 is the break point
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		damp = (0.25);
		A11 = (4.0*damp);
		A12 = (2.0*damp);
		A21 = (1.0*damp);
		A22 = (3.0*damp);
		
		printf("\n********** Matric *********");
		printf("\n %f  %f", A11, A12);
		printf("\n %f  %f", A21, A22);
		printf("\n***************************\n");
	}
	if(key == '1') // Eigenvalue 3/2 eigenvector <1,1> Eigenvalue -1/2 eigenvector <1,-1>
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		damp = (1.0);
		A11 = damp*(1.0/2.0);
		A12 = damp*(1.0);
		A21 = damp*(1.0);
		A22 = damp*(1.0/2.0);
		
		printf("\n********** Matric *********");
		printf("\n %f  %f", A11, A12);
		printf("\n %f  %f", A21, A22);
		printf("\n***************************\n");
	}
	if(key == '2') // Eigenvalue sqrt(3)/2 eigenvector <sqrt(3)+1,1> Eigenvalue -sqrt(3)/2 eigenvector <-sqrt(3)+1,1> equal size eigen values one negative makes it alternate
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		damp = (1.0);
		A11 = damp*(1.0/2.0);
		A12 = damp*(2.0);
		A21 = damp*(1.0/2.0);
		A22 = damp*(-1.0/2.0);
		
		printf("\n********** Matric *********");
		printf("\n %f  %f", A11, A12);
		printf("\n %f  %f", A21, A22);
		printf("\n***************************\n");
	}
	if(key == '3')
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		damp = (1.0);
		A11 = damp*(1.1);
		A12 = damp*(0.0);
		A21 = damp*(0.0);
		A22 = damp*(1.2);
		
		printf("\n********** Matric *********");
		printf("\n %f  %f", A11, A12);
		printf("\n %f  %f", A21, A22);
		printf("\n***************************\n");
	}
	if(key == '4')
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		damp = (1.0);
		A11 = damp*(1.0);
		A12 = damp*(0.0);
		A21 = damp*(0.0);
		A22 = damp*(1.2);
		
		printf("\n********** Matric *********");
		printf("\n %f  %f", A11, A12);
		printf("\n %f  %f", A21, A22);
		printf("\n***************************\n");
	}
	if(key == '5')
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		damp = (1.0);
		float a = 1.1;
		float b = 1.2;
		A11 = damp*(a);
		A12 = damp*(0.0);
		A21 = damp*(b-a);
		A22 = damp*(b);
		
		printf("\n********** Matric *********");
		printf("\n %f  %f", A11, A12);
		printf("\n %f  %f", A21, A22);
		printf("\n***************************\n");
	}
	if(key == '6')
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		damp = (1.0);
		A11 = damp*(0.5);
		A12 = damp*(0.75);
		A21 = damp*(1.0);
		A22 = damp*(0.25);
		
		printf("\n********** Matric *********");
		printf("\n %f  %f", A11, A12);
		printf("\n %f  %f", A21, A22);
		printf("\n***************************\n");
	}
	if(key == '7') // Eigenvalue i eigenvector <-2-i,5> Eigenvalue -i eigenvector <-2+i,5>  spiral 4 cycle
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		damp = (1.0);
		A11 = damp*(2.0);
		A12 = damp*(1.0);
		A21 = damp*(-5.0);
		A22 = damp*(-2.0);
		
		printf("\n********** Matric *********");
		printf("\n %f  %f", A11, A12);
		printf("\n %f  %f", A21, A22);
		printf("\n***************************\n");
	}
	if(key == '8') // Eigenvalue 1/2 + i*sqrt(2)/2 eigenvector <i*sqrt(2),1> Eigenvalue 1/2 - i*sqrt(2)/2 eigenvector <-i*sqrt(2),1>  spiral in
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		damp = (1.0);
		A11 = damp*(1.0/2.0);
		A12 = damp*(-1.0);
		A21 = damp*(1.0/2.0);
		A22 = damp*(1.0/2.0);
		
		printf("\n********** Matric *********");
		printf("\n %f  %f", A11, A12);
		printf("\n %f  %f", A21, A22);
		printf("\n***************************\n");
	}
	if(key == '9') // Eigenvalue damp*(1+i*sqrt(2)) eigenvector <i*sqrt(2),1> Eigenvalue damp*(1-i*sqrt(2)) eigenvector <-i*sqrt(2),1>  spiral out
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		damp = (3.0/5.0);
		A11 = (1.0)*damp;
		A12 = (-2.0)*damp;
		A21 = (1.0)*damp;
		A22 = (1.0)*damp;
		
		printf("\n********** Matric *********");
		printf("\n %f  %f", A11, A12);
		printf("\n %f  %f", A21, A22);
		printf("\n***************************\n");
	}
	if(key == '!') // 
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		printf("\n  Inter start a value\n");
		scanf("%f", &damp);
		
		A11 = (0.0);
		A12 = (-1.0);
		A21 = (1.0);
		A22 = damp;
		
		printf("\n********** Matric *********");
		printf("\n %f  %f", A11, A12);
		printf("\n %f  %f", A21, A22);
		printf("\n***************************\n");
	}
}

void display()
{
	glPointSize(2.0);
	glClear(GL_COLOR_BUFFER_BIT);
	place_axis();
	float damp = (0.25);
	A11 = (4.0*damp);
	A12 = (2.0*damp);
	A21 = (1.0*damp);
	A22 = (3.0*damp);
	glutMouseFunc(mymouse);
}

int main(int argc, char** argv)
{
	glutInit(&argc,argv);
	glutInitWindowSize(X_WINDOW,Y_WINDOW);
	Buffer = new int[X_WINDOW*Y_WINDOW];
	glutInitWindowPosition(0,0);
	g_win = glutCreateWindow("Eigen Values and Eigen Vectors");
	glutKeyboardFunc(KeyPressed);
	glutDisplayFunc(display);
	glutMainLoop();
}
