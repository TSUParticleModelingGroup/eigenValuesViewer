#include "hip/hip_runtime.h"
//nvcc EigenValueViewerSummer25.cu -o temp -lglut -lm -lGLU -lGL
//#include <GL/glut.h>
//#include <math.h>
//#include <stdio.h>
//#include <device.h>

#include <iostream>
#include <fstream>
#include <sstream>
#include <string.h>
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
using namespace std;

#define SCALE 20.0

#define X_WINDOW 1000
#define Y_WINDOW 1000

#define X_MAX SCALE
#define X_MIN -SCALE
#define X_SCALE 1.0

#define Y_MAX SCALE
#define Y_MIN -SCALE
#define Y_SCALE 1.0

FILE* ffmpeg;

//globals
double g_x;
double g_y;
static int g_win;
double A11, A12, A21, A22;
double Real1, Real2, Img1, Img2, EigenMag1, EigenMag2;
double VectorMag1, VectorMag2, Slope1, Slope2;
double Infinity = 100000000.0;
int AdjustA11, AdjustA12, AdjustA21, AdjustA22;
int SingleEigenValue;
int MovieOn = 0;
int* Buffer;
double2 EigenVector1, EigenVector2;
double Discriminate;
double XV1, YV1, XV2, YV2, Projection1, Projection2, Multiplier1, Multiplier2;
double TestX, TestY;

// Prototyping functions
double x_machine_to_x_screen(int);
double y_machine_to_y_screen(int);
double x_machine_to_x_world(int);
double y_machine_to_y_world(int);
double x_world_to_x_screen(double);
double y_world_to_y_screen(double);
void place_axis();
void placePoint(double, double);
void hitMatrix(double, double);
void mymouse(int, int, int, int);
void terminalPrint();
void KeyPressed(unsigned char, int, int);
void display();

double x_machine_to_x_screen(int x)
{
	return( (2.0*x)/X_WINDOW-1.0 );
}

double y_machine_to_y_screen(int y)
{
	return( -(2.0*y)/Y_WINDOW+1.0 );
}

double x_machine_to_x_world(int x)
{
	double range;
	range = X_MAX - X_MIN;
	return( (range/X_WINDOW)*x + X_MIN);
}

double y_machine_to_y_world(int y)
{
	double range;
	range = Y_MAX - Y_MIN;
	return(-((range/Y_WINDOW)*y - X_MAX));
}

double x_world_to_x_screen(double x)
{
	double range;
	range = X_MAX - X_MIN;
	return( -1.0 + 2.0*(x - X_MIN)/range );
}

double y_world_to_y_screen(double y)
{
	double range;
	range = Y_MAX - Y_MIN;
	return( -1.0 + 2.0*(y - Y_MIN)/range );
}

void place_axis()
{
	glColor3f(1.0,1.0,1.0);

	glBegin(GL_LINE_LOOP);
		glVertex2f(x_machine_to_x_screen(0),0);
		glVertex2f(x_machine_to_x_screen(X_WINDOW),0);
	glEnd();

	glBegin(GL_LINE_LOOP);
		glVertex2f(0,y_machine_to_y_screen(0));
		glVertex2f(0,y_machine_to_y_screen(Y_WINDOW));
	glEnd();

	glFlush();
}

void placeEigenVectors()
{
	if(Img1 == 0.0 && Img2 == 0.0)
	{ 
		// Placing eigen vectors
		glColor3f(0.0,1.0,0.0);
		glBegin(GL_LINE_LOOP);
			glVertex2f(x_world_to_x_screen(0.0),y_world_to_y_screen(0.0));
			glVertex2f(x_world_to_x_screen(SCALE*EigenVector1.x/VectorMag1),y_world_to_y_screen(SCALE*EigenVector1.y/VectorMag1));
		glEnd();

		glColor3f(0.0,0.5,0.5);
		glBegin(GL_LINE_LOOP);
			glVertex2f(x_world_to_x_screen(0.0),y_world_to_y_screen(0.0));
			glVertex2f(x_world_to_x_screen(SCALE*EigenVector2.x/VectorMag2),y_world_to_y_screen(SCALE*EigenVector2.y/VectorMag2));
		glEnd();
		
		glColor3f(1.0,0.0,0.0);
		glBegin(GL_LINE_LOOP);
			glVertex2f(x_world_to_x_screen(-SCALE*EigenVector1.x/VectorMag1),y_world_to_y_screen(-SCALE*EigenVector1.y/VectorMag1));
			glVertex2f(x_world_to_x_screen(0.0),y_world_to_y_screen(0.0));
		glEnd();
		
		glColor3f(0.5,0.0,0.5);
		glBegin(GL_LINE_LOOP);
			glVertex2f(x_world_to_x_screen(-SCALE*EigenVector2.x/VectorMag2),y_world_to_y_screen(-SCALE*EigenVector2.y/VectorMag2));
			glVertex2f(x_world_to_x_screen(0.0),y_world_to_y_screen(0.0));
		glEnd();
		
		// Placing dots magnitude of eigen value
		glColor3f(1.0,0.0,0.0);
		glPointSize(5);
		glBegin(GL_POINTS);
			glVertex2f(x_world_to_x_screen(EigenMag1*EigenVector1.x/VectorMag1),y_world_to_y_screen(EigenMag1*EigenVector1.y/VectorMag1));
			glVertex2f(x_world_to_x_screen(EigenMag2*EigenVector2.x/VectorMag2),y_world_to_y_screen(EigenMag2*EigenVector2.y/VectorMag2));
		glEnd();
		
		// Placing one unit on eigen vector
		glColor3f(1.0,1.0,1.0);
		glBegin(GL_LINE_LOOP);
			glVertex2f(x_world_to_x_screen(0.0),y_world_to_y_screen(0.0));
			glVertex2f(x_world_to_x_screen(EigenVector1.x/VectorMag1),y_world_to_y_screen(EigenVector1.y/VectorMag1));
		glEnd();

		glBegin(GL_LINE_LOOP);
			glVertex2f(x_world_to_x_screen(0.0),y_world_to_y_screen(0.0));
			glVertex2f(x_world_to_x_screen(EigenVector2.x/VectorMag2),y_world_to_y_screen(EigenVector2.y/VectorMag2));
		glEnd();
	}

	glFlush();
}

void placePoint(double x, double y)
{
	glColor3f(1.0,0.0,1.0);
	glBegin(GL_POINTS);
		glVertex2f(x_world_to_x_screen(TestX),y_world_to_y_screen(TestY));
	glEnd();
	glFlush();
	
	glColor3f(0.0,1.0,1.0);
	glBegin(GL_POINTS);
		glVertex2f(x_world_to_x_screen(XV1),y_world_to_y_screen(YV1));
	glEnd();
	glFlush();
	
	glColor3f(0.0,0.5,1.0);
	glBegin(GL_POINTS);
		glVertex2f(x_world_to_x_screen(XV2),y_world_to_y_screen(YV2));
	glEnd();
	glFlush();
	
	if(MovieOn == 1)
	{
		glReadPixels(5, 5, X_WINDOW, Y_WINDOW, GL_RGBA, GL_UNSIGNED_BYTE, Buffer);
		fwrite(Buffer, sizeof(int)*X_WINDOW*Y_WINDOW, 1, ffmpeg);
	}
}

void hitMatrix(double x, double y)
{
	double xOld = x;
	double yOld = y;
	
	g_x = A11*xOld + A12*yOld;
	g_y = A21*xOld + A22*yOld;
}

void findValues()
{
	Discriminate = (A11 - A22)*(A11 - A22) + 4.0*(A12*A21);
	
	if(0.0 < Discriminate)
	{
		Real1 = 0.5*(A11 + A22 - sqrt(Discriminate));
		Real2 = 0.5*(A11 + A22 + sqrt(Discriminate));
		Img1 = 0.0;
		Img2 = 0.0;
		SingleEigenValue = 0;
	}
	else if(0.0 == Discriminate)
	{
		Real1 = 0.5*(A11 + A22);
		Real2 = 0.5*(A11 + A22);
		Img1 = 0.0;
		Img2 = 0.0;
		SingleEigenValue = 1;
	}
	else
	{
		Real1 = 0.5*(A11 + A22);
		Real2 = 0.5*(A11 + A22);
		Img1 = -0.5*sqrt(-Discriminate);
		Img2 = +0.5*sqrt(-Discriminate);
		SingleEigenValue = 0;
	}
	EigenMag1 = sqrt(Img1*Img1 + Real1*Real1);
	EigenMag2 = sqrt(Img2*Img2 + Real2*Real2);
	
	if(A12 != 0.0)
	{
		EigenVector1.x = 1.0;
		EigenVector1.y = (A22 - A11 - sqrt(Discriminate))/(2.0*A12);
		
		EigenVector2.x = 1.0;
		EigenVector2.y = (A22 - A11 + sqrt(Discriminate))/(2.0*A12);
	}
	else
	{
		if((A11 - A22) != 0.0)
		{
			EigenVector1.x = 1.0;
			EigenVector1.y = A21/(A11 - A22);
			
			EigenVector2.x = 0.0;
			EigenVector2.y = 1.0;
		}
		else
		{
			EigenVector1.x = 0.0;
			EigenVector1.y = 1.0;
			
			EigenVector2.x = 0.0;
			EigenVector2.y = 1.0;
		}
		
	}
	VectorMag1 = sqrt(EigenVector1.x*EigenVector1.x + EigenVector1.y*EigenVector1.y);
	VectorMag2 = sqrt(EigenVector2.x*EigenVector2.x + EigenVector2.y*EigenVector2.y);
	
	if(EigenVector1.x != 0.0)
	{
		Slope1 = EigenVector1.y/EigenVector1.x;
	}
	else
	{
		Slope1 = Infinity + 1.0;
	}
	
	if(EigenVector2.x != 0.0)
	{
		Slope2 = EigenVector2.y/EigenVector2.x;
	}
	else
	{
		Slope2 = Infinity + 1.0;
	}
}

void terminalPrint()
{
	system("clear");
	
	if(AdjustA11)
	{
		printf("\n  \033[1;32mA11\033[0m  A12   %f  %f", A11, A12);
		printf("\n  A21  A22   %f  %f", A21, A22);
	}
	else if(AdjustA12)
	{
		printf("\n  A11  \033[1;32mA12\033[0m   %f  %f", A11, A12);
		printf("\n  A21  A22   %f  %f", A21, A22);
	}
	else if(AdjustA21)
	{
		printf("\n  A11  A12   %f  %f", A11, A12);
		printf("\n  \033[1;32mA21\033[0m  A22   %f  %f", A21, A22);
	}
	else if(AdjustA22)
	{
		printf("\n  A11  A12   %f  %f", A11, A12);
		printf("\n  A21  \033[1;32mA22\033[0m   %f  %f", A21, A22);
	}
	else
	{
		printf("\n  A11  A12   %f  %f", A11, A12);
		printf("\n  A21  A22   %f  %f", A21, A22);
	}
	printf("\n");
	
	
	printf("\n  EigenValue1 =   %f + %fi: mag = %f", Real1, Img1, EigenMag1);
	printf("\n  EigenValue2 =   %f + %fi: mag = %f", Real2, Img2, EigenMag2);
	printf("\n  (A11 - A22)^2 + 4.0*(A12*A21) Discriminate =   %f", Discriminate);
	printf("\n");
	
	printf("\n  EigenVector1 =   <%f, %f>", EigenVector1.x, EigenVector1.y);
	printf("\n  EigenVector2 =   <%f, %f>", EigenVector2.x, EigenVector2.y);
	printf("\n");
	
	printf("\n  Slope1 = %f", Slope1);
	printf("\n  Slope2 = %f", Slope2);
	printf("\n");
	
	printf("\n  x = %f\n",g_x);
	printf("  y = %f\n",g_y);
	
	printf("\n  x test = %f\n",TestX);
	printf("  y test = %f\n",TestY);
	
	printf("\n");
}

void mymouse(int button, int state, int x, int y)
{	
	if(state == GLUT_DOWN)
	{
		if(button == GLUT_LEFT_BUTTON)
		{
			g_x = x_machine_to_x_world(x);
			g_y = y_machine_to_y_world(y);
			
			glColor3f(1.0,1.0,0.0);
			glPointSize(5);
			glBegin(GL_POINTS);
				glVertex2f(x_world_to_x_screen(g_x),y_world_to_y_screen(g_y));
			glEnd();
			glFlush();
			
			TestX = g_x;
			TestY = g_y;
			
			if(Infinity <= Slope1)
			{
				XV1 = 0.0;
				if(Infinity <= Slope2)
				{
					YV1 = TestY;
				}
				else
				{
					YV1 = -Slope2*TestX + TestY;
				}
				Projection1 = YV1;
				
			}
			else if(Slope1 != 0.0)
			{
				if(Infinity <= Slope2)
				{
					XV1 = TestX;
					YV1 = Slope1*XV1;
					Projection1 = sqrt(XV1*XV1 + YV1*YV1);
					if(XV1*EigenVector1.x < 0.0) Projection1 = -Projection1;
				}
				else
				{
					XV1 = ((-Slope2)*TestX + TestY)/(Slope1 - Slope2);
					YV1 = Slope1*XV1;
					Projection1 = sqrt(XV1*XV1 + YV1*YV1);
					if(XV1*EigenVector1.x < 0.0) Projection1 = -Projection1;
				}
			}
			else
			{
				XV1 = TestX;
				YV1 = 0.0;
				Projection1 = XV1;
			}
			
			if(Infinity <= Slope2)
			{
				XV2 = 0.0;
				if(Infinity <= Slope1)
				{
					YV2 = TestY;
				}
				else
				{
					YV2 = -Slope1*TestX + TestY;
				}
				Projection2 = YV2;
			}
			else if(Slope2 != 0.0)
			{
				XV2 = ((-Slope1)*TestX + TestY)/(Slope2 - Slope1);
				YV2 = Slope2*XV2;
				Projection2 = sqrt(XV2*XV2 + YV2*YV2);
				if(XV2*EigenVector2.x < 0.0) Projection2 = -Projection2;
			}
			else
			{
				XV2 = TestX;
				YV2 = 0.0;
				Projection2 = XV2;
			}
			
			Multiplier1 = Real1;
			Multiplier2 = Real2;
			
			// This should go right on top of the original point.
			glColor3f(1.0,0.0,1.0);
			glBegin(GL_POINTS);
				glVertex2f(x_world_to_x_screen(TestX),y_world_to_y_screen(TestY));
			glEnd();
			glFlush();
			
			// This should be the projection of the point on to the first vector.
			glColor3f(0.0,1.0,1.0);
			glBegin(GL_POINTS);
				glVertex2f(x_world_to_x_screen(XV1),y_world_to_y_screen(YV1));
			glEnd();
			glFlush();
			
			// This should be the projection of the point on to the second vector.
			glColor3f(0.0,0.5,1.0);
			glBegin(GL_POINTS);
				glVertex2f(x_world_to_x_screen(XV2),y_world_to_y_screen(YV2));
			glEnd();
			glFlush();
	
			//placePoint(g_x,g_y);
			terminalPrint();
		}
		else
		{
			hitMatrix(g_x,g_y);
			
			glColor3f(0.0,1.0,0.0);
			glPointSize(5);
			glBegin(GL_POINTS);
				glVertex2f(x_world_to_x_screen(g_x),y_world_to_y_screen(g_y));
			glEnd();
			glFlush();
			
			if(Infinity <= Slope1)
			{
				XV1 = 0.0;
				YV1 = Multiplier1*Projection1;
			}
			else if(Slope1 != 0.0)
			{
				XV1 = Multiplier1*Projection1*EigenVector1.x/VectorMag1;
				YV1 = Multiplier1*Projection1*EigenVector1.y/VectorMag1;
			}
			else
			{
				XV1 = Multiplier1*Projection1;
				YV1 = 0.0;
			}
			
			if(Infinity <= Slope2)
			{
				XV2 = 0.0;
				YV2 = Multiplier2*Projection2;
			}
			if(Slope2 != 0.0)
			{
				XV2 = Multiplier2*Projection2*EigenVector2.x/VectorMag2;
				YV2 = Multiplier2*Projection2*EigenVector2.y/VectorMag2;
			}
			else
			{
				XV2 = Multiplier2*Projection2;
				YV2 = 0.0;
			}
			
			// This should be the projection of the point on to the first vector.
			glColor3f(0.0,1.0,1.0);
			glBegin(GL_POINTS);
				glVertex2f(x_world_to_x_screen(XV1),y_world_to_y_screen(YV1));
			glEnd();
			glFlush();
			
			// This should be the projection of the point on to the second vector.
			glColor3f(0.0,0.5,1.0);
			glBegin(GL_POINTS);
				glVertex2f(x_world_to_x_screen(XV2),y_world_to_y_screen(YV2));
			glEnd();
			glFlush();
			
			TestX = XV1 + XV2;
			TestY = YV1 + YV2;
			
			Multiplier1 *= Real1;
			Multiplier2 *= Real2;
	
			//placePoint(g_x,g_y);
			terminalPrint();
		}
	}
}

void KeyPressed(unsigned char key, int x, int y)
{
	float tempx;
	float tempy;
	float damp;
	if(key == 'q')
	{
		glutDestroyWindow(g_win);
		exit(0);
	}
	if(key == 'k')
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		placeEigenVectors();
	}
	
	if(key == 't')
	{
		printf("\n  Inter start x value\n");
		scanf("%f", &tempx);
		g_x = tempx;
		printf("\n  Inter start y value\n");
		scanf("%f", &tempy);
		g_y = tempy;
		
		glColor3f(1.0,0.0,0.0);
		//placePoint(g_x,g_y);
		terminalPrint();
	}
	
	if(key == 'a')
	{
		AdjustA11 = 1;
		AdjustA12 = 0;
		AdjustA21 = 0;
		AdjustA22 = 0;
		terminalPrint();
	}
	if(key == 'b')
	{
		AdjustA11 = 0;
		AdjustA12 = 1;
		AdjustA21 = 0;
		AdjustA22 = 0;
		terminalPrint();
	}
	if(key == 'c')
	{
		AdjustA11 = 0;
		AdjustA12 = 0;
		AdjustA21 = 1;
		AdjustA22 = 0;
		terminalPrint();
	}
	if(key == 'd')
	{
		AdjustA11 = 0;
		AdjustA12 = 0;
		AdjustA21 = 0;
		AdjustA22 = 1;
		terminalPrint();
	}
	
	double adjustment = 0.01;
	if(key == '+')
	{
		if(AdjustA11 == 1)      A11 += adjustment;
		else if(AdjustA12 == 1) A12 += adjustment; 
		else if(AdjustA21 == 1) A21 += adjustment; 
		else if(AdjustA22 == 1) A22 += adjustment; 
		findValues();
		terminalPrint();
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		placeEigenVectors();
	}
	if(key == '-')
	{
		if(AdjustA11 == 1)      A11 -= adjustment;
		else if(AdjustA12 == 1) A12 -= adjustment; 
		else if(AdjustA21 == 1) A21 -= adjustment; 
		else if(AdjustA22 == 1) A22 -= adjustment;
		findValues();
		terminalPrint();
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		placeEigenVectors(); 
	}
	
	if(key == 'm')
	{
		// Setting up the movie buffer.
		const char* cmd = "ffmpeg -r 60 -f rawvideo -pix_fmt rgba -s 1000x1000 -i - "
		              "-threads 0 -preset fast -y -pix_fmt yuv420p -crf 21 -vf vflip output.mp4";
		ffmpeg = popen(cmd, "w");
		//Buffer = new int[XWindowSize*YWindowSize];
		Buffer = (int*)malloc(X_WINDOW*Y_WINDOW*sizeof(int));
		MovieOn = 1;
	}
	if(key == 'M')
	{
		pclose(ffmpeg);
		MovieOn = 0;
	}
	
	if(key == 's')
	{	
		FILE* ScreenShotFile;
		int* buffer;
		const char* cmd = "ffmpeg -r 60 -f rawvideo -pix_fmt rgba -s 1000x1000 -i - "
		              "-threads 0 -preset fast -y -pix_fmt yuv420p -crf 21 -vf vflip output1.mp4";
		ScreenShotFile = popen(cmd, "w");
		buffer = (int*)malloc(X_WINDOW*Y_WINDOW*sizeof(int));
		
		for(int i =0; i < 1; i++)
		{
			glReadPixels(5, 5, X_WINDOW, Y_WINDOW, GL_RGBA, GL_UNSIGNED_BYTE, buffer);
			fwrite(buffer, sizeof(int)*X_WINDOW*Y_WINDOW, 1, ScreenShotFile);
		}
		
		pclose(ScreenShotFile);
		free(buffer);
		system("ffmpeg -i output1.mp4 screenShot.jpeg");
		system("rm output1.mp4");
	}
	
	if(key == 'h')
	{
		printf("\n q: Quit");
		printf("\n k: Clear");
		printf("\n a: Adjust A11");
		printf("\n b: Adjust A12");
		printf("\n c: Adjust A21");
		printf("\n d: Adjust A22");
		printf("\n +/-: Adjust up/down");
		printf("\n M/m: Movie on/off");
		printf("\n s: Screen Shot");
		printf("\n");
		printf("\n t: Read starting x and y from screen.");
		printf("\n 0: Eigenvalue 5/4 eigenvector <2,1> Eigenvalue 1/2 eigenvector <1,-1>");
		printf("\n 1: Eigenvalue 3/2 eigenvector <1,1> Eigenvalue -1/2 eigenvector <1,-1>");
		printf("\n 2: Eigenvalue sqrt(5)/2 eigenvector <sqrt(5)+1,1> Eigenvalue -sqrt(3)/2 eigenvector <-sqrt(3)+1,1> equal size eigen values one negative makes it alternate");
		printf("\n 3: Eigenvalue 1.1 eigenvector <1,0> Eigenvalue 1.2 eigenvector <0,1>");
		printf("\n 4: Eigenvalue 1 eigenvector <1,0> Eigenvalue 1.2 <0,1>");
		printf("\n 5: Eigenvalue 1.1 eigenvector <1,1> Eigenvalue 1.2 eigenvector <0,1>");
		printf("\n 6: Eigenvalue -0.5 eigenvector <-3,4> Eigenvalue 1.25 eigenvector <1,1>");
		printf("\n 7: Eigenvalue i eigenvector <-2-i,5> Eigenvalue -i eigenvector <-2+i,5>  spiral 4 cycle");
		printf("\n 8: Eigenvalue 1/2 + i*sqrt(2)/2 eigenvector <i*sqrt(2),1> Eigenvalue 1/2 - i*sqrt(2)/2 eigenvector <-i*sqrt(2),1>  spiral in");
		printf("\n 9: Eigenvalue damp*(1+i*sqrt(2)) eigenvector <i*sqrt(2),1> Eigenvalue damp*(1-i*sqrt(2)) eigenvector <-i*sqrt(2),1>  spiral out");
		printf("\n");
	}
	
	if(key == '0') // Eigenvalue 5/4 eigenvector <2,1> Eigenvalue 1/2 eigenvector <1,-1> .2 is the break point
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		damp = (0.25);
		A11 = (4.0*damp);
		A12 = (2.0*damp);
		A21 = (1.0*damp);
		A22 = (3.0*damp);
		findValues();
		terminalPrint();
		placeEigenVectors();
	}
	if(key == '1') // Eigenvalue 3/2 eigenvector <1,1> Eigenvalue -1/2 eigenvector <1,-1>
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		damp = (1.0);
		A11 = damp*(1.0/2.0);
		A12 = damp*(1.0);
		A21 = damp*(1.0);
		A22 = damp*(1.0/2.0);
		findValues();
		terminalPrint();
		placeEigenVectors();
	}
	if(key == '2') // Eigenvalue sqrt(3)/2 eigenvector <sqrt(3)+1,1> Eigenvalue -sqrt(3)/2 eigenvector <-sqrt(3)+1,1> equal size eigen values one negative makes it alternate
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		damp = (1.0);
		A11 = damp*(1.0/2.0);
		A12 = damp*(2.0);
		A21 = damp*(1.0/2.0);
		A22 = damp*(-1.0/2.0);
		findValues();
		terminalPrint();
		placeEigenVectors();
	}
	if(key == '3')
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		damp = (1.0);
		A11 = damp*(1.1);
		A12 = damp*(0.0);
		A21 = damp*(0.0);
		A22 = damp*(1.2);
		findValues();
		terminalPrint();
		placeEigenVectors();
	}
	if(key == '4')
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		damp = (1.0);
		A11 = damp*(1.0);
		A12 = damp*(0.0);
		A21 = damp*(0.0);
		A22 = damp*(1.2);
		findValues();
		terminalPrint();
		placeEigenVectors();
	}
	if(key == '5')
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		damp = (1.0);
		float a = 1.1;
		float b = 1.2;
		A11 = damp*(a);
		A12 = damp*(0.0);
		A21 = damp*(b-a);
		A22 = damp*(b);
		findValues();
		terminalPrint();
		placeEigenVectors();
	}
	if(key == '6')
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		damp = (1.0);
		A11 = damp*(0.5);
		A12 = damp*(0.75);
		A21 = damp*(1.0);
		A22 = damp*(0.25);
		findValues();
		terminalPrint();
		placeEigenVectors();
	}
	if(key == '7') // Eigenvalue i eigenvector <-2-i,5> Eigenvalue -i eigenvector <-2+i,5>  spiral 4 cycle
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		damp = (1.0);
		A11 = damp*(2.0);
		A12 = damp*(1.0);
		A21 = damp*(-5.0);
		A22 = damp*(-2.0);
		findValues();
		terminalPrint();
		placeEigenVectors();
	}
	if(key == '8') // Eigenvalue 1/2 + i*sqrt(2)/2 eigenvector <i*sqrt(2),1> Eigenvalue 1/2 - i*sqrt(2)/2 eigenvector <-i*sqrt(2),1>  spiral in
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		damp = (1.0);
		A11 = damp*(1.0/2.0);
		A12 = damp*(-1.0);
		A21 = damp*(1.0/2.0);
		A22 = damp*(1.0/2.0);
		findValues();
		terminalPrint();
		placeEigenVectors();
	}
	if(key == '9') // Eigenvalue damp*(1+i*sqrt(2)) eigenvector <i*sqrt(2),1> Eigenvalue damp*(1-i*sqrt(2)) eigenvector <-i*sqrt(2),1>  spiral out
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		damp = (3.0/5.0);
		A11 = (1.0)*damp;
		A12 = (-2.0)*damp;
		A21 = (1.0)*damp;
		A22 = (1.0)*damp;
		findValues();
		terminalPrint();
		placeEigenVectors();
	}
	if(key == '!') // 
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		
		A11 = (0.0);
		A12 = (2.0);
		A21 = (3.0);
		A22 = (0.0);
		findValues();
		terminalPrint();
		placeEigenVectors();
	}
}

void display()
{
	glPointSize(2.0);
	glClear(GL_COLOR_BUFFER_BIT);
	place_axis();
	float damp = (0.25);
	A11 = (4.0*damp);
	A12 = (2.0*damp);
	A21 = (1.0*damp);
	A22 = (3.0*damp);
	findValues();
	terminalPrint();
	placeEigenVectors();
	glutMouseFunc(mymouse);
}

int main(int argc, char** argv)
{
	glutInit(&argc,argv);
	glutInitWindowSize(X_WINDOW,Y_WINDOW);
	Buffer = new int[X_WINDOW*Y_WINDOW];
	glutInitWindowPosition(0,0);
	g_win = glutCreateWindow("Eigen Values and Eigen Vectors");
	glutKeyboardFunc(KeyPressed);
	glutDisplayFunc(display);
	glutMainLoop();
}
