//nvcc EigenValueViewerICTCM.cu -o temp -lglut -lm -lGLU -lGL
//#include <GL/glut.h>
//#include <math.h>
//#include <stdio.h>
//#include <device.h>

#include <iostream>
#include <fstream>
#include <sstream>
#include <string.h>
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
using namespace std;

#define SCALE 50.0

#define X_WINDOW 1000
#define Y_WINDOW 1000

#define X_MAX SCALE
#define X_MIN -SCALE
#define X_SCALE 1.0

#define Y_MAX SCALE
#define Y_MIN -SCALE
#define Y_SCALE 1.0

FILE* ffmpeg;

// function prototypes
void KeyPressed(unsigned char key, int x, int y);
void Display(void);

//globalsgcc FunctionHit.c -o FunctionHit -lglut -lm -lGLU -lGL
double g_x;
double g_y;
static int g_win;
double A11, A12, A21, A22;
int* Buffer;

double x_machine_to_x_screen(int x)
{
	return( (2.0*x)/X_WINDOW-1.0 );
}

double y_machine_to_y_screen(int y)
{
	return( -(2.0*y)/Y_WINDOW+1.0 );
}

double x_machine_to_x_world(int x)
{
	double range;
	range = X_MAX - X_MIN;
	return( (range/X_WINDOW)*x + X_MIN);
}

double y_machine_to_y_world(int y)
{
	double range;
	range = Y_MAX - Y_MIN;
	return(-((range/Y_WINDOW)*y - X_MAX));
}

double x_world_to_x_screen(double x)
{
	double range;
	range = X_MAX - X_MIN;
	return( -1.0 + 2.0*(x - X_MIN)/range );
}

double y_world_to_y_screen(double y)
{
	double range;
	range = Y_MAX - Y_MIN;
	return( -1.0 + 2.0*(y - Y_MIN)/range );
}

void place_axis()
{
	glColor3f(1.0,1.0,1.0);

	glBegin(GL_LINE_LOOP);
		glVertex2f(x_machine_to_x_screen(0),0);
		glVertex2f(x_machine_to_x_screen(X_WINDOW),0);
	glEnd();

	glBegin(GL_LINE_LOOP);
		glVertex2f(0,y_machine_to_y_screen(0));
		glVertex2f(0,y_machine_to_y_screen(Y_WINDOW));
	glEnd();

	glFlush();
}

void placePoint(double x, double y)
{
	glPointSize(5);
	glBegin(GL_POINTS);
		glVertex2f(x_world_to_x_screen(x),y_world_to_y_screen(y));
	glEnd();
	glFlush();
}

void hitMatrix(double x, double y)
{
	double xOld = x;
	double yOld = y;
	
	g_x = A11*xOld + A12*yOld;
	g_y = A21*xOld + A22*yOld;
}

void printPoint()
{
	printf("\n  x = %f\n",g_x);
	printf("  y = %f\n",g_y);
}

void mymouse(int button, int state, int x, int y)
{	
	if(state == GLUT_DOWN)
	{
		if(button == GLUT_LEFT_BUTTON)
		{
			glColor3f(1.0,1.0,0.0);
			g_x = x_machine_to_x_world(x);
			g_y = y_machine_to_y_world(y);
			placePoint(g_x,g_y);
			printPoint();
		}
		else
		{
			glColor3f(0.0,1.0,0.0);
			hitMatrix(g_x,g_y);
			placePoint(g_x,g_y);
			printPoint();
		}
	}
}

void help()
{
	printf("\n Click in the black x-y axis then hit one of the following options.");
	printf("\n After sellecting options (1,3,4,5,6) left click the mouse to set your");
	printf("\n initial condition then right click the mouse to generate your next point.");
	printf("\n 1: Sets up the matrix that generated figure 1 in the paper.");
	printf("\n 3: Sets up the matrix that generated figure 3 in the paper.");
	printf("\n 4: Sets up the matrix that generated figure 4 in the paper.");
	printf("\n 5: Sets up the matrix that generated figure 5 in the paper.");
	printf("\n 6: Sets up the matrix that generated figure 6 in the paper.");
	printf("\n n: Allows the user to enter their own 2X2 matrix. Note you will enter these values in the linux terminal");
	printf("\n s: Takes a screan shot.");
	printf("\n c: Clears the screan.");
	printf("\n h: Displays the help screan again.");
	printf("\n q: Quits the program.");
}

void KeyPressed(unsigned char key, int x, int y)
{
	if(key == 'q')
	{
		glutDestroyWindow(g_win);
		exit(0);
	}
	
	if(key == 'c')
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
	}
	
	if(key == 's')
	{	
		FILE* ScreenShotFile;
		int* buffer;
		const char* cmd = "ffmpeg -r 60 -f rawvideo -pix_fmt rgba -s 1000x1000 -i - "
		              "-threads 0 -preset fast -y -pix_fmt yuv420p -crf 21 -vf vflip output1.mp4";
		ScreenShotFile = popen(cmd, "w");
		buffer = (int*)malloc(X_WINDOW*Y_WINDOW*sizeof(int));
		
		for(int i =0; i < 1; i++)
		{
			glReadPixels(5, 5, X_WINDOW, Y_WINDOW, GL_RGBA, GL_UNSIGNED_BYTE, buffer);
			fwrite(buffer, sizeof(int)*X_WINDOW*Y_WINDOW, 1, ScreenShotFile);
		}
		
		pclose(ScreenShotFile);
		free(buffer);
		system("ffmpeg -i output1.mp4 screenShot.jpeg");
		system("rm output1.mp4");
	}
	
	if(key == 'h')
	{
		help();
	}
	
	if(key == '1') // Produces figure 1
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		A11 = 1.0;
		A12 = 1.0/2.0;
		A21 = 1.0/4.0;
		A22 = 3.0/4.0;
	}
	if(key == '3') // Produces figure 3
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		A11 = 1.0/2.0;
		A12 = 3.0/4.0;
		A21 = 1.0;
		A22 = 1.0/4.0;
	}
	if(key == '4') // Produces figure 
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		A11 = 1.1;
		A12 = 0.0;
		A21 = 0.0;
		A22 = 1.2;
	}
	if(key == '5') // Produces figure 5
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		A11 = 1.0;
		A12 = 0.0;
		A21 = 0.0;
		A22 = 1.2;
	}
	if(key == '6') // Produces figure 6
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		A11 = 1.0/2.0;
		A12 = 2.0;
		A21 = 1.0/2.0;
		A22 = -1.0/2.0;
	}
	if(key == 'n')
	{
		glClear(GL_COLOR_BUFFER_BIT);
		place_axis();
		printf("\n  Enter A11\n");
		scanf("%lf", &A11);
		printf("\n  Enter A12\n");
		scanf("%lf", &A12);
		printf("\n  Enter A21\n");
		scanf("%lf", &A21);
		printf("\n  Enter A22\n");
		scanf("%lf", &A22);
		printf("\n  Done. Go back to the and click on the x-y screan.\n");
	}
}

void display()
{
	glPointSize(2.0);
	glClear(GL_COLOR_BUFFER_BIT);
	place_axis();
	glutMouseFunc(mymouse);
}

int main(int argc, char** argv)
{
	glutInit(&argc,argv);
	glutInitWindowSize(X_WINDOW,Y_WINDOW);
	Buffer = new int[X_WINDOW*Y_WINDOW];
	glutInitWindowPosition(0,0);
	g_win = glutCreateWindow("Eigen Values and Eigen Vectors");
	glutKeyboardFunc(KeyPressed);
	glutDisplayFunc(display);
	help();
	glutMainLoop();
}
